#include "hip/hip_runtime.h"
#include <algorithm>

#include "boost/cuda/types.hpp"
#include "boost/cuda/assert.h"
#include "externals/cxx/utility.hpp"

#include "cuda/matrix.hpp"
#include "cuda/kernel/reduce.hpp"

template<typename T>
__global__
static void kernel(const rysq::matrix_data_array<T> A, T scale, rysq::matrix<T> B) {
    T r = 0;
    int i  = threadIdx.x + blockIdx.x* blockDim.x;
    int j = blockIdx.y;
    int ij = A. layout().element_at(i,j);
    if (i <  A.layout().size1) {
	for (int k = 0; k < A.size(); ++k) {
	    r += A[k][ij];
	}  
	B(i,j) += scale*r;
    }
}

template<typename T>
static void reduce_(const rysq::matrix_data_array<T> A,
		    T scale, rysq::matrix<T> B,
		    hipStream_t stream) {
    size_t size1 = A.layout().size1, size2 = A.layout().size2;
    dim3 block(std::min<size_t>(size1, 128), 1, 1);
    dim3 grid(cxx::utility::qceiling<size_t>(size1, block.x), size2, 1);
    // std::cout <<  block << grid << " " << size1 << std::endl;
    // std:: cout << block << grid << A.size() << std::endl;
    ::kernel<<< grid, block, 0, stream >>>(A, scale, B); 
    //cuda_assert();
}

void rysq::cuda::reduce(const rysq::matrix_data_array<double> A,
			double scale, rysq::matrix<double> B,
			hipStream_t stream) {
    ::reduce_<double>(A, scale, B, stream);
}

